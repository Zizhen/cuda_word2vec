
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

// Compile with:
// nvcc -o example example.cu
#define N 1000

__global__
void add(int *a, int *b) {
    int i = blockIdx.x;
    if (i<N) {
        b[i] = 2*a[i];
    }
}

int main() {
    int ha[N], hb[N];

    int *da, *db;
    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));

    for (int i = 0; i<N; ++i) {
        ha[i] = i;
    }

    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);

    add<<<N, 1>>>(da, db);

    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i<N; ++i) {
      cout << hb[i] << endl;
    }

    hipFree(da);
    hipFree(db);

    return 0;
}
