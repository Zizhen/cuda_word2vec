#include <iostream>
#include <string.h>
#include <fstream>
#include <vector>
#include <sstream>
#include <unordered_map>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

// Compile with:
// nvcc -o example example.cu
#define N 10

#define ERROR_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__
void add(int *a, int *b) {
  int i = blockIdx.x;
  if (i<N) {
    b[i] = 2*a[i];
  }
}

int main(int argc, char* argv[]) {
  if(argc == 2){
    if(strcmp(argv[1], "analogy") == 0)
      cout << "usage: ./a.out analogy path/to/your/model path/to/your/testfile dimension" << endl;
    else
      cout << "function not supported" << endl;
  }
  else if (argc > 2){
    size_t f, t;
    hipSetDevice(0);
    hipMemGetInfo(&f, &t);
    cout << f << " " << t << endl;

    unordered_map<string, int> word2vec_map;
    string str;
    ifstream infile;
    infile.open(argv[2]);
    int word_count = 0;
    int dimension = stod(argv[4]);
    while(getline(infile,str)){
      word_count++;
    }
    // cout << word_count << " " << dimension << endl;
    int matrix_size = word_count*dimension;
    cout << matrix_size << endl;
    float word_matrix_h[matrix_size];
    float* word_matrix_d;
    ERROR_CHECK(hipMalloc((void **)&word_matrix_d, 10693650*sizeof(float)));
    hipError_t err = hipMalloc((void **)&word_matrix_d, matrix_size*sizeof(float));
    if(err == hipErrorOutOfMemory){
      cout << "error" << endl;
    } else{
      cout << "you are good" << endl;
    }
    hipSetDevice(0);
    hipMemGetInfo(&f, &t);
    cout << f << " " << t << endl;

    // cout << matrix_size << endl;

    int i = 0;
    while(getline(infile,str)){
      string buf;
      stringstream ss(str);
      ss >> buf;
      cout << buf << endl;
      word2vec_map[buf] = i;
      int j = 0;
      while (ss >> buf){
        cout << buf << endl;
        word_matrix_h[i*dimension+j] = stod(buf);
        j++;
      }
      i++;
    }
    infile.close();

    // for (int i = 0; i < 10; i++){
    //   for (int j = 0; j < 10; j++){
    //     cout << word_matrix[i*dimension+j] << " ";
    //   }
    //   cout << endl;
    // }

    hipFree(word_matrix_d);

  }
  ////////////////////////////////
    int ha[N], hb[N];
    int *da, *db;
    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));

    for (int i = 0; i<N; ++i) {
        ha[i] = i;
    }

    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);

    add<<<N, 1>>>(da, db);

    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i<N; ++i) {
      cout << hb[i] << endl;
    }

    hipFree(da);
    hipFree(db);

    return 0;
}
