#include <iostream>
#include <string.h>
#include <fstream>
#include <vector>
#include <sstream>
#include <unordered_map>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <algorithm>
#include <math.h>
#include "hipblas.h"

using namespace std;

int main(){
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  int m = 3;
  int n = 4;
  int k = 2;
  float *A_h = new float[6];
  A_h[0] = 1;
  A_h[1] = 2;
  A_h[2] = 3;
  A_h[3] = 2;
  A_h[4] = 3;
  A_h[5] = 4;
  float *B_h = new float[8];
  B_h[0] = 1;
  B_h[1] = 4;
  B_h[2] = 2;
  B_h[3] = 3;
  B_h[4] = 3;
  B_h[5] = 2;
  B_h[6] = 4;
  B_h[7] = 1;
  float *C_h = new float[12];
  float *A_d, *B_d, *C_d;
  hipMalloc((void **)&A_d, 6*sizeof(float));
  hipMalloc((void **)&B_d, 8*sizeof(float));
  hipMalloc((void **)&C_d, 12*sizeof(float));
  hipMemcpy(A_d, A_h, 6*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, 8*sizeof(float), hipMemcpyHostToDevice);
  const float alpha = 1.0f;
  const float beta = 0.0f;
  hipblasSgemm(
    handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
    m, n, k,
    &alpha,
    A_d, m,
    B_d, k,
    &beta,
    C_d, m
  );
  hipMemcpy(C_h, C_d, 12*sizeof(float), hipMemcpyDeviceToHost);
  for(int i = 0; i < 12; i ++){
    cout << C_h[i] << endl;
  }

  return 1;
}
