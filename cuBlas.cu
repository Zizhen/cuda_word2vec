#include <iostream>
#include <string.h>
#include <fstream>
#include <vector>
#include <sstream>
#include <unordered_map>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <algorithm>
#include <math.h>
#include "hipblas.h"

using namespace std;

// Compile with:
// nvcc -o example example.cu
#define ERROR_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__
void normalize(float* mat, float* normSum_d, float* matrixNorm_d, int dim){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  for(int j = 0; j < dim; j++){
    matrixNorm_d[i*dim+j] = mat[i*dim+j] / normSum_d[i];
  }
}

__global__
void vectorManipulation(float* A, float* B, float* C, float* D, int len){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < len)
    D[i] = A[i] + C[i] - B[i];
}

__global__
void vecMatMultiplication(float* mat, float* vec, float* res, int len, int max){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if(i < max){
    for(int j = 0; j < len; j ++){
      res[i] += mat[i*len+j] * vec[j];
    }
  }
}

int main(int argc, char* argv[]) {
  if(argc == 2){
    if(strcmp(argv[1], "analogy") == 0){
      cout << "usage: ./a.out analogy path/to/your/model dim path/to/your/testfile" << endl;
      cout << "   or: ./a.out analogy path/to/your/model dim word1 word2 word3" << endl;
    }
    else
      cout << "function not supported" << endl;
  }
  else if (argc > 2){
    // size_t f, t;
    // cudaSetDevice(0);
    // cudaMemGetInfo(&f, &t);
    // cout << f << " " << t << endl;

    unordered_map<string, int> word2vec_map;
    string str;
    ifstream infile;
    infile.open(argv[2]);
    int word_count = 0;
    int dim = stod(argv[3]);
    hipblasHandle_t handle;

    while(getline(infile,str)){
      word_count++;
    }
    infile.close();
    string dictionary[word_count];
    float normSum_h[word_count];

    infile.open(argv[2]);
    int matrix_size = word_count*dim;
    float *matrix_h = new float[matrix_size];
    float *resVec_h = new float[word_count];
    int i = 0;
    while(getline(infile,str)){
      string buf;
      stringstream ss(str);
      ss >> buf;
      word2vec_map[buf] = i;
      dictionary[i] = buf;
      int j = 0;
      while (ss >> buf){
        matrix_h[i*dim+j] = stof(buf);
        normSum_h[i] += pow(stof(buf),2);
        j++;
      }
      normSum_h[i] = sqrt(normSum_h[i]);
      i++;
    }
    infile.close();

    // no need for change up to here
    hipblasCreate(&handle);
    float* matrix_d;
    float* matrixNorm_d;
    float* predict_d;
    float* resVec_d;
    float* normSum_d;

    hipMalloc((void **)&matrix_d, matrix_size*sizeof(float));
    hipMalloc((void **)&matrixNorm_d, matrix_size*sizeof(float));
    hipMalloc((void **)&predict_d, dim*sizeof(float));
    hipMalloc((void **)&resVec_d, word_count*sizeof(float));
    hipMalloc((void **)&normSum_d, word_count*sizeof(float));

    hipMemcpy(matrix_d, matrix_h, matrix_size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(normSum_d, normSum_h, word_count*sizeof(float), hipMemcpyHostToDevice);

    // cublasSetMatrix (word_count, dim, sizeof(float), matrix_h, word_count, matrix_d, word_count);
    // cublasSetMatrix (word_count, 1, sizeof(float), normSum_h, word_count, normSum_d, word_count);

    dim3 dimGrid(ceil(word_count/1024.0), 1, 1);
    dim3 dimBlock(1024, 1, 1);
    normalize<<<dimGrid, dimBlock>>>(matrix_d, normSum_d, matrixNorm_d, dim);

    if(strcmp(argv[1],"analogy") == 0){
      if(argc == 7){
        int count[3];
        for(int i = 0; i < 3; i++){
          count[i] = word2vec_map.count(argv[4+i]);
          if(count[i] != 1){
              cout << "map does not contain the word: " << argv[4+i] << endl;
              return -1;
          }
        }
        int idx_1 = word2vec_map[argv[4]];
        int idx_2 = word2vec_map[argv[5]];
        int idx_3 = word2vec_map[argv[6]];
        dim3 dimGrid1(1, 1, 1);
        dim3 dimBlock1(dim, 1, 1);
        vectorManipulation<<<dimGrid1, dimBlock1>>>(&matrixNorm_d[idx_1*dim],
                  &matrixNorm_d[idx_2*dim], &matrixNorm_d[idx_3*dim], predict_d, dim);

        const float alpha = 1.0f;
        const float beta = 0.0f;
        cout << "test 1" << endl;
        hipblasSgemm(
          handle,
          HIPBLAS_OP_N,
          HIPBLAS_OP_N,
          word_count, 1, dim,
          &alpha,
          matrixNorm_d, word_count,
          predict_d, dim,
          &beta,
          resVec_d, word_count
        );
        cout << "test 2" << endl;

        hipMemcpy(resVec_h, resVec_d, word_count*sizeof(float), hipMemcpyDeviceToHost);
        resVec_h[idx_1] = 0;
        resVec_h[idx_2] = 0;
        resVec_h[idx_3] = 0;
        int max = std::max_element(resVec_h, resVec_h + word_count) - resVec_h;
        cout << dictionary[max] << endl;
      }
    }

    hipblasDestroy(handle);
    hipFree(matrix_d);
    hipFree(matrixNorm_d);
    hipFree(predict_d);
    hipFree(resVec_d);
  }

  return 0;
}
