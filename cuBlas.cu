#include <iostream>
#include <string.h>
#include <fstream>
#include <vector>
#include <sstream>
#include <unordered_map>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <algorithm>
#include <math.h>
#include "hipblas.h"

using namespace std;

// Compile with:
// nvcc -o example example.cu
#define ERROR_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__
void normalize_T(float* mat, float* normSum_d, float* matrixNorm_d, float* matrixNorm_T, int word_count, int dim){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if(i < word_count){
    for(int j = 0; j < dim; j++){
      matrixNorm_d[i*dim+j] = mat[i*dim+j] / normSum_d[i];
      matrixNorm_T[j*word_count+i] = mat[i*dim+j] / normSum_d[i];
    }
  }
}

__global__
void vectorManipulation(float* A, float* B, float* C, float* D, int len){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < len)
    D[i] = A[i] + C[i] - B[i];
}

__global__
void vecMatMultiplication(float* mat, float* vec, float* res, int len, int max){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if(i < max){
    for(int j = 0; j < len; j ++){
      res[i] += mat[i*len+j] * vec[j];
    }
  }
}

int main(int argc, char* argv[]) {
  if(argc == 2){
    if(strcmp(argv[1], "analogy") == 0){
      cout << "usage: ./a.out analogy path/to/your/model dim path/to/your/testfile" << endl;
      cout << "   or: ./a.out analogy path/to/your/model dim word1 word2 word3" << endl;
    }
    else
      cout << "function not supported" << endl;
  }
  else if (argc > 2){
    // size_t f, t;
    // cudaSetDevice(0);
    // cudaMemGetInfo(&f, &t);
    // cout << f << " " << t << endl;
    float elapsed=0;
    hipEvent_t start, stop;
    ERROR_CHECK(hipEventCreate(&start));
    ERROR_CHECK(hipEventCreate(&stop));

    unordered_map<string, int> word2vec_map;
    string str;
    ifstream infile;
    infile.open(argv[2]);
    int word_count = 0;
    int dim = stod(argv[3]);
    hipblasHandle_t handle;

    while(getline(infile,str)){
      word_count++;
    }
    infile.close();
    string dictionary[word_count];
    float normSum_h[word_count];

    infile.open(argv[2]);
    int matrix_size = word_count*dim;
    float *matrix_h = new float[matrix_size];
    float *resVec_h = new float[word_count];
    int i = 0;
    while(getline(infile,str)){
      string buf;
      stringstream ss(str);
      ss >> buf;
      word2vec_map[buf] = i;
      dictionary[i] = buf;
      int j = 0;
      while (ss >> buf){
        matrix_h[i*dim+j] = stof(buf);
        normSum_h[i] += pow(stof(buf),2);
        j++;
      }
      normSum_h[i] = sqrt(normSum_h[i]);
      i++;
    }
    infile.close();

    // no need for change up to here
    hipblasCreate(&handle);
    float* matrix_d;
    float* matrixNorm_d;
    float* matrixNorm_T;
    float* predict_d;
    float* resVec_d;
    float* normSum_d;

    ERROR_CHECK(hipMalloc((void **)&matrix_d, matrix_size*sizeof(float)));
    ERROR_CHECK(hipMalloc((void **)&matrixNorm_d, matrix_size*sizeof(float)));
    ERROR_CHECK(hipMalloc((void **)&matrixNorm_T, matrix_size*sizeof(float)));
    ERROR_CHECK(hipMalloc((void **)&predict_d, dim*sizeof(float)));
    ERROR_CHECK(hipMalloc((void **)&resVec_d, word_count*sizeof(float)));
    ERROR_CHECK(hipMalloc((void **)&normSum_d, word_count*sizeof(float)));

    ERROR_CHECK(hipMemcpy(matrix_d, matrix_h, matrix_size*sizeof(float), hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpy(normSum_d, normSum_h, word_count*sizeof(float), hipMemcpyHostToDevice));

    dim3 dimGrid(ceil(word_count/1024.0), 1, 1);
    dim3 dimBlock(1024, 1, 1);
    normalize_T<<<dimGrid, dimBlock>>>(matrix_d, normSum_d, matrixNorm_d, matrixNorm_T, word_count, dim);
    ERROR_CHECK(hipDeviceSynchronize());

    if(strcmp(argv[1],"analogy") == 0){
      if(argc == 7){
        int count[3];
        for(int i = 0; i < 3; i++){
          count[i] = word2vec_map.count(argv[4+i]);
          if(count[i] != 1){
              cout << "map does not contain the word: " << argv[4+i] << endl;
              return -1;
          }
        }
        int idx_1 = word2vec_map[argv[4]];
        int idx_2 = word2vec_map[argv[5]];
        int idx_3 = word2vec_map[argv[6]];
        dim3 dimGrid1(1, 1, 1);
        dim3 dimBlock1(dim, 1, 1);
        vectorManipulation<<<dimGrid1, dimBlock1>>>(&matrixNorm_d[idx_1*dim],
                  &matrixNorm_d[idx_2*dim], &matrixNorm_d[idx_3*dim], predict_d, dim);

        ERROR_CHECK(hipDeviceSynchronize());
        const float alpha = 1.0f;
        const float beta = 0.0f;

        ERROR_CHECK(hipEventRecord(start, 0));
        hipblasSgemm(
          handle,
          HIPBLAS_OP_N,
          HIPBLAS_OP_N,
          word_count, 1, dim,
          &alpha,
          matrixNorm_T, word_count,
          predict_d, dim,
          &beta,
          resVec_d, word_count
        );
        ERROR_CHECK(hipEventRecord(stop, 0));
        ERROR_CHECK(hipEventSynchronize(stop));
        ERROR_CHECK(hipEventElapsedTime(&elapsed, start, stop));
        ERROR_CHECK(hipEventDestroy(start));
        ERROR_CHECK(hipEventDestroy(stop));
        cout << "Vector-matrix multiplication takes " << elapsed << " ms" << endl;

        ERROR_CHECK(hipMemcpy(resVec_h, resVec_d, word_count*sizeof(float), hipMemcpyDeviceToHost));

        resVec_h[idx_1] = 0;
        resVec_h[idx_2] = 0;
        resVec_h[idx_3] = 0;
        int max = std::max_element(resVec_h, resVec_h + word_count) - resVec_h;
        cout << dictionary[max] << endl;
      }
    }

    hipblasDestroy(handle);
    ERROR_CHECK(hipFree(matrix_d));
    ERROR_CHECK(hipFree(matrixNorm_d));
    ERROR_CHECK(hipFree(matrixNorm_T));
    ERROR_CHECK(hipFree(predict_d));
    ERROR_CHECK(hipFree(resVec_d));
    ERROR_CHECK(hipFree(normSum_d));

  }

  return 0;
}
