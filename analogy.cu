#include <iostream>
#include <string.h>
#include <fstream>
#include <vector>
#include <sstream>
#include <unordered_map>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <algorithm>
#include <math.h>
#include "hipblas.h"

using namespace std;

// Compile with:
// nvcc -o example example.cu
#define ERROR_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__
void normalize_T(float* mat, float* normSum_d, float* matrixNorm_d, float* matrixNorm_T, int word_count, int dim){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if(i < word_count){
    for(int j = 0; j < dim; j++){
      matrixNorm_d[i*dim+j] = mat[i*dim+j] / normSum_d[i];
      matrixNorm_T[j*word_count+i] = mat[i*dim+j] / normSum_d[i];
    }
  }
}

__global__
void vectorManipulation(float* A, float* B, float* C, float* D, int len){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < len)
    D[i] = A[i] + C[i] - B[i];
}

__global__
void fillABC(float* A_d, float* B_d, float* C_d, int* A_idx, int* B_idx, int* C_idx, float* matrixNorm_d, int len, int dim){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < len){
    memcpy(&A_d[i*dim], &matrixNorm_d[A_idx[i]*dim], sizeof(float)*dim);
    memcpy(&B_d[i*dim], &matrixNorm_d[B_idx[i]*dim], sizeof(float)*dim);
    memcpy(&C_d[i*dim], &matrixNorm_d[C_idx[i]*dim], sizeof(float)*dim);
  }
}

__global__
void correctness(float* resVec_d, int* A_idx_d, int* B_idx_d, int* C_idx_d, int* D_idx_d, int word_count, int query_count){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < query_count){
    int idx_a = A_idx_d[i];
    int idx_b = B_idx_d[i];
    int idx_c = C_idx_d[i];
    int max_idx = 0;
    int max = -1;
    for(int j = 0; j < word_count; j ++){
      if(j == idx_a || j == idx_b || j == idx_c)
        continue;
      if(resVec_d[i*word_count + j] > max){
        max = resVec_d[i*word_count + j];
        max_idx = j;
      }
    }
    D_idx_d[i] = max_idx;
  }
}

void fill_host_vector(string filename, unordered_map<string, int> &word2vec_map, string *dictionary, float *normSum_h, float *matrix_h, int dim){
  string str;
  ifstream infile;
  infile.open(filename);
  int i = 0;
  while(getline(infile,str)){
    string buf;
    stringstream ss(str);
    ss >> buf;
    word2vec_map[buf] = i;
    dictionary[i] = buf;
    int j = 0;
    while (ss >> buf){
      matrix_h[i*dim+j] = stof(buf);
      normSum_h[i] += pow(stof(buf),2);
      j++;
    }
    normSum_h[i] = sqrt(normSum_h[i]);
    i++;
  }
  infile.close();
}

std::tuple<int, int> find_dim_wordcount(string filename){
  int word_count = 0;
  string str;
  ifstream infile;
  infile.open(filename);
  getline(infile,str);
  word_count++;
  stringstream sss(str);
  int dim = -1;
  string buff;
  while(sss >> buff)
    dim ++;
  while(getline(infile,str)){
    word_count++;
  }
  infile.close();
  return std::make_tuple(dim, word_count);
}

int fill_host_query_vector(string queryFile, unordered_map<string, int> word2vec_map, int *A_idx, int *B_idx, int *C_idx, int *D_idx){
  string str;
  ifstream infile;
  infile.open(queryFile);
  int i = 0;
  int model_miss = 0;
  while(getline(infile, str)){
    string buf;
    stringstream ss(str);
    vector<string> queryWords;
    while (ss >> buf){
      if (buf[0] >= 'A' and buf[0] <= 'Z')
        buf[0] = buf[0]-'A'+'a';
      queryWords.push_back(buf);
    }
    int occurence_count;
    for(int j = 0; j < 4; j++){
      occurence_count = word2vec_map.count(queryWords[j]);
      if(occurence_count != 1){
        model_miss += 1;
        continue;
      }
    }
    A_idx[i] = word2vec_map[queryWords[1]];
    B_idx[i] = word2vec_map[queryWords[0]];
    C_idx[i] = word2vec_map[queryWords[2]];
    D_idx[i] = word2vec_map[queryWords[3]];
    i++;
  }
  infile.close();
  cout << "model doesn't cover " << model_miss << " of the total queries" << endl;
  return model_miss;
}

int analogy_single_query(string filename, vector<string> queryWords) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int dim, word_count;
    tie(dim, word_count) = find_dim_wordcount(filename);
    int matrix_size = word_count*dim;

    unordered_map<string, int> word2vec_map;
    string dictionary[word_count];
    float *normSum_h = new float[word_count];
    float *matrix_h = new float[matrix_size];
    float *resVec_h = new float[word_count];
    fill_host_vector(filename, word2vec_map, dictionary, normSum_h, matrix_h, dim);

    // check if all query words are in model
    int occurence_count;
    for(int i = 0; i < 3; i++){
      occurence_count = word2vec_map.count(queryWords[i]);
      if(occurence_count != 1){
        cout << "model does not contain the word: " << queryWords[i] << endl;
        return -1;
      }
    }

    float* matrix_d;
    float* matrixNorm_d;
    float* matrixNorm_T;
    float* predict_d;
    float* resVec_d;
    float* normSum_d;

    ERROR_CHECK(hipMalloc((void **)&matrix_d, matrix_size*sizeof(float)));
    ERROR_CHECK(hipMalloc((void **)&matrixNorm_d, matrix_size*sizeof(float)));
    ERROR_CHECK(hipMalloc((void **)&matrixNorm_T, matrix_size*sizeof(float)));
    ERROR_CHECK(hipMalloc((void **)&predict_d, dim*sizeof(float)));
    ERROR_CHECK(hipMalloc((void **)&resVec_d, word_count*sizeof(float)));
    ERROR_CHECK(hipMalloc((void **)&normSum_d, word_count*sizeof(float)));

    ERROR_CHECK(hipMemcpy(matrix_d, matrix_h, matrix_size*sizeof(float), hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpy(normSum_d, normSum_h, word_count*sizeof(float), hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemset(predict_d, 0.0, dim*sizeof(float)));

    dim3 dimGrid(ceil(word_count/1024.0), 1, 1);
    dim3 dimBlock(1024, 1, 1);
    normalize_T<<<dimGrid, dimBlock>>>(matrix_d, normSum_d, matrixNorm_d, matrixNorm_T, word_count, dim);
    ERROR_CHECK(hipDeviceSynchronize());

    int idx_1 = word2vec_map[queryWords[0]];
    int idx_2 = word2vec_map[queryWords[1]];
    int idx_3 = word2vec_map[queryWords[2]];

    const float alpha = 1.0f;
    const float beta = 0.0f;
    dim3 dimGrid1(1, 1, 1);
    dim3 dimBlock1(dim, 1, 1);
    vectorManipulation<<<dimGrid1, dimBlock1>>>(&matrixNorm_d[idx_1*dim],
              &matrixNorm_d[idx_2*dim], &matrixNorm_d[idx_3*dim], predict_d, dim);
    ERROR_CHECK(hipDeviceSynchronize());

    hipblasSgemv(
      handle, HIPBLAS_OP_N,
      word_count, dim,
      &alpha, matrixNorm_T, std::max(word_count, dim),
      predict_d, 1, &beta,
      resVec_d, 1
    );

    ERROR_CHECK(hipMemcpy(resVec_h, resVec_d, word_count*sizeof(float), hipMemcpyDeviceToHost));

    resVec_h[idx_1] = 0;
    resVec_h[idx_2] = 0;
    resVec_h[idx_3] = 0;
    int max = std::max_element(resVec_h, resVec_h + word_count) - resVec_h;
    cout << dictionary[max] << endl;

    hipblasDestroy(handle);
    ERROR_CHECK(hipFree(matrix_d));
    ERROR_CHECK(hipFree(matrixNorm_d));
    ERROR_CHECK(hipFree(matrixNorm_T));
    ERROR_CHECK(hipFree(predict_d));
    ERROR_CHECK(hipFree(resVec_d));
    ERROR_CHECK(hipFree(normSum_d));
    delete [] normSum_h;
    delete [] matrix_h;
    delete [] resVec_h;

    return 0;
}

int analogy_batch_query(string filename, string queryFile) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int dim, word_count, query_count;
    tie(dim, query_count) = find_dim_wordcount(queryFile);
    tie(dim, word_count) = find_dim_wordcount(filename);
    int matrix_size = word_count*dim;

    unordered_map<string, int> word2vec_map;
    string dictionary[word_count];
    float *normSum_h = new float[word_count];
    float *matrix_h = new float[matrix_size];
    int *A_idx = new int[query_count];
    int *B_idx = new int[query_count];
    int *C_idx = new int[query_count];
    int *D_idx = new int[query_count];
    int *D_res = new int[query_count];
    fill_host_vector(filename, word2vec_map, dictionary, normSum_h, matrix_h, dim);
    query_count -= fill_host_query_vector(queryFile, word2vec_map, A_idx, B_idx, C_idx, D_idx);
    int query_matrix_size = query_count*dim;

    float* matrix_d;
    float* matrixNorm_d;
    float* matrixNorm_T;
    float* predict_d;
    float* resVec_d;
    float* normSum_d;
    int* A_idx_d;
    int* B_idx_d;
    int* C_idx_d;
    int* D_idx_d;
    float* A_d;
    float* B_d;
    float* C_d;

    ERROR_CHECK(hipMalloc((void **)&matrix_d, matrix_size*sizeof(float)));
    ERROR_CHECK(hipMalloc((void **)&matrixNorm_d, matrix_size*sizeof(float)));
    ERROR_CHECK(hipMalloc((void **)&matrixNorm_T, matrix_size*sizeof(float)));
    ERROR_CHECK(hipMalloc((void **)&predict_d, query_matrix_size*sizeof(float)));
    ERROR_CHECK(hipMalloc((void **)&resVec_d, word_count*query_count*sizeof(float)));
    ERROR_CHECK(hipMalloc((void **)&normSum_d, word_count*sizeof(float)));
    ERROR_CHECK(hipMalloc((void **)&A_idx_d, query_count*sizeof(int)));
    ERROR_CHECK(hipMalloc((void **)&B_idx_d, query_count*sizeof(int)));
    ERROR_CHECK(hipMalloc((void **)&C_idx_d, query_count*sizeof(int)));
    ERROR_CHECK(hipMalloc((void **)&D_idx_d, query_count*sizeof(int)));
    ERROR_CHECK(hipMalloc((void **)&A_d, query_matrix_size*sizeof(float)));
    ERROR_CHECK(hipMalloc((void **)&B_d, query_matrix_size*sizeof(float)));
    ERROR_CHECK(hipMalloc((void **)&C_d, query_matrix_size*sizeof(float)));

    ERROR_CHECK(hipMemcpy(matrix_d, matrix_h, matrix_size*sizeof(float), hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpy(normSum_d, normSum_h, word_count*sizeof(float), hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpy(A_idx_d, A_idx, query_count*sizeof(int), hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpy(B_idx_d, B_idx, query_count*sizeof(int), hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpy(C_idx_d, C_idx, query_count*sizeof(int), hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemset(predict_d, 0.0, query_matrix_size*sizeof(float)));

    dim3 dimGrid(ceil(word_count/1024.0), 1, 1);
    dim3 dimBlock(1024, 1, 1);
    normalize_T<<<dimGrid, dimBlock>>>(matrix_d, normSum_d, matrixNorm_d, matrixNorm_T, word_count, dim);
    ERROR_CHECK(hipDeviceSynchronize());

    // in case fillABC does not work
    for(int i = 0; i < query_count; i++){
      ERROR_CHECK(hipMemcpy(&A_d[i*dim], &matrixNorm_d[A_idx[i]*dim], dim*sizeof(float), hipMemcpyDeviceToDevice));
      ERROR_CHECK(hipMemcpy(&B_d[i*dim], &matrixNorm_d[B_idx[i]*dim], dim*sizeof(float), hipMemcpyDeviceToDevice));
      ERROR_CHECK(hipMemcpy(&C_d[i*dim], &matrixNorm_d[C_idx[i]*dim], dim*sizeof(float), hipMemcpyDeviceToDevice));
    }

    const float alpha = 1.0f;
    const float negative_alpha = -1.0f;
    const float beta = 0.0f;
    hipblasSaxpy(
      handle, query_matrix_size,
      &alpha,
      A_d, 1,
      predict_d, 1
    );
    ERROR_CHECK(hipDeviceSynchronize());
    hipblasSaxpy(
      handle, query_matrix_size,
      &negative_alpha,
      B_d, 1,
      predict_d, 1
    );
    ERROR_CHECK(hipDeviceSynchronize());
    hipblasSaxpy(
      handle, query_matrix_size,
      &alpha,
      C_d, 1,
      predict_d, 1
    );
    ERROR_CHECK(hipDeviceSynchronize());

    hipblasSgemm(
      handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
      word_count, query_count, dim,
      &alpha,
      matrixNorm_T, word_count,
      predict_d, dim,
      &beta,
      resVec_d, word_count
    );
    ERROR_CHECK(hipDeviceSynchronize());

    float* resVec_h = new float[query_count*word_count];
    ERROR_CHECK(hipMemcpy(resVec_h, resVec_d, query_count*word_count*sizeof(float), hipMemcpyDeviceToHost));

    int total_correct = 0;
    float* resVec_tmp;
    for(int i = 0; i < query_count; i++){
      int idx_a = A_idx[i];
      int idx_b = B_idx[i];
      int idx_c = C_idx[i];
      resVec_tmp = &resVec_h[i*word_count];
      resVec_tmp[idx_a] = 0;
      resVec_tmp[idx_b] = 0;
      resVec_tmp[idx_c] = 0;
      int max = std::max_element(resVec_tmp, resVec_tmp + word_count) - resVec_tmp;

      if(max == D_idx[i]){
        total_correct ++;
      }
    }
    cout << "total_correct is: " << total_correct << " out of: " << query_count << endl;
    cout << "Correctness is: " << (total_correct*1.0)/query_count << endl;

    hipblasDestroy(handle);
    ERROR_CHECK(hipFree(matrix_d));
    ERROR_CHECK(hipFree(matrixNorm_d));
    ERROR_CHECK(hipFree(matrixNorm_T));
    ERROR_CHECK(hipFree(predict_d));
    ERROR_CHECK(hipFree(resVec_d));
    ERROR_CHECK(hipFree(normSum_d));
    delete [] normSum_h;
    delete [] matrix_h;
    delete [] resVec_h;
    delete [] A_idx;
    delete [] B_idx;
    delete [] C_idx;
    delete [] D_idx;
    delete [] D_res;

    return 0;
}
